#include "hip/hip_runtime.h"
#include "pic_cuvol_render.cuh"
#include "render_util.cuh"

namespace {

__device__ __inline__ void render_background_backward(
            const svox2::device::PackedSparseGridSpec& __restrict__ grid,
            const float* __restrict__ grad_output,
            svox2::device::SingleRaySpec& __restrict__ ray,
            const RenderOptions& __restrict__ opt,
            float log_transmit,
            float accum,
            float sparsity_loss,
            svox2::device::PackedGridOutputGrads& __restrict__ grads
        ) {
    // printf("accum_init=%f\n", accum);
    // printf("log_transmit_init=%f\n", log_transmit);
    svox2::device::ConcentricSpheresIntersector csi(ray.origin, ray.dir);

    const int n_steps = int(grid.background_nlayers / opt.step_size) + 2;

    const float inner_radius = fmaxf(_dist_ray_to_origin(ray.origin, ray.dir) + 1e-3f, 1.f);
    float t, invr_last = 1.f / inner_radius;
    // csi.intersect(inner_radius, &t_last);
    for (int i = 0; i < n_steps; ++i) {
        float r = n_steps / (n_steps - i - 0.5);

        if (r < inner_radius || !csi.intersect(r, &t)) continue;

#pragma unroll 3
        for (int j = 0; j < 3; ++j) {
            ray.pos[j] = fmaf(t, ray.dir[j], ray.origin[j]);
        }

        const float invr_mid = _rnorm(ray.pos);
#pragma unroll 3
        for (int j = 0; j < 3; ++j) {
            ray.pos[j] *= invr_mid;
        }
        // NOTE: reusing ray.pos (ok if you check _unitvec2equirect)
        svox2::device::_unitvec2equirect(ray.pos, grid.background_reso, ray.pos);
        ray.pos[2] = fminf(fmaxf((1.f - invr_mid) * grid.background_nlayers - 0.5f, 0.f),
                       grid.background_nlayers - 1);
#pragma unroll 3
        for (int j = 0; j < 3; ++j) {
            ray.l[j] = (int) ray.pos[j];
        }
        ray.l[0] = min(ray.l[0], grid.background_reso * 2 - 1);
        ray.l[1] = min(ray.l[1], grid.background_reso - 1);
        ray.l[2] = min(ray.l[2], grid.background_nlayers - 2);
#pragma unroll 3
        for (int j = 0; j < 3; ++j) {
            ray.pos[j] -= ray.l[j];
        }


        float sigma = svox2::device::trilerp_bg_one(
                grid.background_links,
                grid.background_data,
                grid.background_reso,
                grid.background_nlayers,
                4,
                ray.l,
                ray.pos,
                3);
        // if (i == n_steps - 1) {
        //     ray.world_step = 1e9;
        // }

        // if (opt.randomize && opt.random_sigma_std_background > 0.0)
        //     sigma += ray.rng.randn() * opt.random_sigma_std_background;
        if (sigma > 0.f) {
            float total_color = 0.f;
            const float pcnt = ray.world_step * (invr_last - invr_mid) * sigma;
            const float weight = _EXP(log_transmit) * (1.f - _EXP(-pcnt));
            log_transmit -= pcnt;

            for (int i = 0; i < 3; ++i) {
                const float color = svox2::device::trilerp_bg_one(
                        grid.background_links,
                        grid.background_data,
                        grid.background_reso,
                        grid.background_nlayers,
                        4,
                        ray.l,
                        ray.pos,
                        i) * svox2::device::C0 + 0.5f;  // Scale by SH DC factor to help normalize lrs

                total_color += fmaxf(color, 0.f) * grad_output[i];
                if (color > 0.f) {
                    const float curr_grad_color = svox2::device::C0 * weight * grad_output[i];
                    svox2::device::trilerp_backward_bg_one(
                            grid.background_links,
                            grads.grad_background_out,
                            nullptr,
                            grid.background_reso,
                            grid.background_nlayers,
                            4,
                            ray.l,
                            ray.pos,
                            curr_grad_color,
                            i);
                }
            }

            accum -= weight * total_color;
            float curr_grad_sigma = ray.world_step * (invr_last - invr_mid) * (
                    total_color * _EXP(log_transmit) - accum);
            if (sparsity_loss > 0.f) {
                // Cauchy version (from SNeRG)
                curr_grad_sigma += sparsity_loss * (4 * sigma / (1 + 2 * (sigma * sigma)));

                // Alphs version (from PlenOctrees)
                // curr_grad_sigma += sparsity_loss * _EXP(-pcnt) * ray.world_step;
            }

            svox2::device::trilerp_backward_bg_one(
                    grid.background_links,
                    grads.grad_background_out,
                    grads.mask_background_out,
                    grid.background_reso,
                    grid.background_nlayers,
                    4,
                    ray.l,
                    ray.pos,
                    curr_grad_sigma,
                    3);

            if (_EXP(log_transmit) < opt.stop_thresh) {
                break;
            }
        }
        invr_last = invr_mid;
    }
}

} // end of namespace

__launch_bounds__(svox2::TRACE_RAY_BG_CUDA_THREADS, svox2::MIN_BG_BLOCKS_PER_SM)
__global__ void pic::render_background_backward_kernel(
    svox2::device::PackedSparseGridSpec grid,
    const float* __restrict__ grad_output,
    const float* __restrict__ color_cache,
    svox2::device::PackedRaysSpec rays,
    RenderOptions opt,
    const float* __restrict__ log_transmit,
    const float* __restrict__ accum,
    bool grad_out_is_rgb,
    float sparsity_loss,
    // Outputs
    svox2::device::PackedGridOutputGrads grads) {
    CUDA_GET_THREAD_ID(ray_id, int(rays.origins.size(0)));
    if (log_transmit[ray_id] < -25.f) return;
    svox2::device::SingleRaySpec ray_spec(rays.origins[ray_id].data(), rays.dirs[ray_id].data());
    ray_find_bounds_bg(ray_spec, grid, opt, ray_id);

    float grad_out[3];
    if (grad_out_is_rgb) {
        const float norm_factor = 2.f / (3 * int(rays.origins.size(0)));
#pragma unroll 3
        for (int i = 0; i < 3; ++i) {
            const float resid = color_cache[ray_id * 3 + i] - grad_output[ray_id * 3 + i];
            grad_out[i] = resid * norm_factor;
        }
    } else {
#pragma unroll 3
        for (int i = 0; i < 3; ++i) {
            grad_out[i] = grad_output[ray_id * 3 + i];
        }
    }

    render_background_backward(
        grid,
        grad_out,
        ray_spec,
        opt,
        log_transmit[ray_id],
        accum[ray_id],
        sparsity_loss,
        grads);
}